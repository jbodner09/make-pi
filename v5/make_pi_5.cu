/*This program calculates pi using a Simpson's Rule estimation of the
integral of arctangent from 0 to 1. When inputting the number of 
iterations to perform, more iterations = more precision. The number of
iterations is given as a command line argument. If no argument is 
provided, a default value of 20,000 is used. At 20,000 iterations, the
value of pi is guaranteed to be accurate up to 8 decimal places. This
version uses NVIDIA CUDA to perform parallel computation of the 
partial sums on a GPU.  

The amount of work each core does is given by the two #defines below. 
These values will need to be tuned for each device this code runs on in
order to get maximum performance. For example, on the Oakley cluster of
the Ohio Supercomputer Center, which uses the Tesla M2070, there are 
14 streaming multiprocessors (SM's), with 32 cores each, for a total of
448 cores. Thus, 448 threads will be created, with each thread performing
multiple iterations (total_iterations / (NUM_BLOCKS * THREADS_PER_BLOCK)
to be precise). Thus, the more iterations given, the more work each thread
does. The number of threads is kept constant in order to make clean-up 
easier and to not exceed the capabilities (max number of threads or blocks)
of any particular GPU device. The last thread might have a few extra 
iterations if that number doesn't divide evenly. 

The number of decimal digits to use as the precision of the calculations is
also given as a command-line argument. Obviously, the higher the number, the   
more digits you can successfully calculate. Accuracy still relies on the number
of iterations, though: a high number of digits but low number of iterations
will still result in a low number of digits of precision. Thus, you should
only increase the number of digits when your iterations get too high and 
you find that your calculations are no longer precise due to internal 
rounding error. You'll probably find that increasing the digits will decrease
performance severely. It is recommended, though, that since error accumulates,
the more digits you want to find, the more padding you'll need to add to the
end of the word to absorb that error. As a general rule of thumb, if you 
want to calculate x digits, make your words 2x long. Of course, this also
increases the runtime by 2x. 

Compilation on OSC is a little complicated. First, log into oakley.osc.edu and
request an interactive computation node since it has the GPU: 
    qsub -I -l walltime=0:59:00 -l nodes=1:gpus=1
Once you have acquired a node, first load the CUDA module: 
    module load cuda
Then compile it with the NVIDIA compiler: 
    nvcc -O -arch sm_20 -o make_pi_5 make_pi_5.cu
Finally, to run it, just give it the iterations and precision arguments:
    ./make_pi_5 20000 25
The -arch option is needed because we use dynamic memory allocation on the GPU.
Only second-generation and higher CUDA devices support this, so this code won't
run on the oldest NVIDIA graphics cards. 
*/

// Includes. Optimum values for OSC are:
// NUM_BLOCKS 14
// THREADS_PER_BLOCK 32

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>

#define NUM_BLOCKS 14
#define THREADS_PER_BLOCK 32

// A bignum is stored as all its decimal digits, separated into an array.
// Really, it's quite terrible for performance, but it allows infinite digits.
// Or at least as many as we can store in memory. The power tells us where to
// put the decimal point, and the number of significant digits tells us how
// many of the digits in the number are actually used. The precision tells us
// the maximum number of digits possible for this particular instance.
typedef struct {
    signed long int power;
    unsigned long int sig_digs;
    char * digits;
    unsigned long int precision;
} bignum;

// Function pointers, mostly for bignum operations. Note that in our use
// below, we assume most of the arithmetic functions don't fail and thus 
// don't check their return values. Hope they're tested well... Notice
// now that we have mirrored versions for the GPU, most of which just
// have to call the GPU memory allocation functions. 
__global__ void calculate (long *, long *, char *, long *, long *, char *, long, long);
__host__ bignum * bignum_init(long int);
__host__ void bignum_reset(bignum *);
__host__ void bignum_clear(bignum *);
__host__ int bignum_set_int(bignum *, long int);
__host__ void bignum_set(bignum *, bignum *);
__host__ void bignum_print(bignum *, long int);
__host__ int bignum_add(bignum *, bignum *, bignum *);
__host__ int bignum_add_int(bignum *, bignum *, long int);
__host__ int bignum_mult(bignum *, bignum *, bignum *);
__host__ int bignum_mult_int(bignum *, bignum *, long int);
__host__ int bignum_divide(bignum *, bignum *, bignum *);
__host__ int bignum_int_divide(bignum *, long int, bignum *);
__host__ int bignum_divide_int(bignum *, bignum *, long int);
__device__ bignum * bignum_init_gpu(long int);
__device__ void bignum_reset_gpu(bignum *);
__device__ void bignum_clear_gpu(bignum *);
__device__ int bignum_set_int_gpu(bignum *, long int);
__device__ void bignum_set_gpu(bignum *, bignum *);
__device__ int bignum_add_gpu(bignum *, bignum *, bignum *);
__device__ int bignum_add_int_gpu(bignum *, bignum *, long int);
__device__ int bignum_mult_gpu(bignum *, bignum *, bignum *);
__device__ int bignum_mult_int_gpu(bignum *, bignum *, long int);
__device__ int bignum_divide_gpu(bignum *, bignum *, bignum *);
__device__ int bignum_int_divide_gpu(bignum *, long int, bignum *);
__device__ int bignum_divide_int_gpu(bignum *, bignum *, long int);

// Main function
int main (int argc, char * argv[])
{
    // Obtain command line arguments
    long iterations = 20000L;
    if (argc > 1) {
        iterations = atol (argv[1]);
        if (iterations < 1L) {
            iterations = 20000L;
        }
    }
    long max_digits = 25L;
    if (argc > 2) {
        max_digits = atoi (argv[2]);
        if (max_digits < 1L) {
            max_digits = 25L;
        }
    }
    
    // Initialize global storage. Notice that we now need extra arrays for data 
    // transfer between the GPU and regular RAM. These will hold the partial 
    // sums that each of the threads calculate. Unfortunately, due to the way
    // bignums are structured, each of their arguments has to be transferred 
    // separately. Luckily, this only happens once. 
    long clock_start = (long)clock();
    long int i, j;
    if (hipDeviceSetLimit(hipLimitMallocHeapSize, (NUM_BLOCKS * THREADS_PER_BLOCK * 16384))
        != hipSuccess) { printf("\nError setting GPU heap size.\n"); return 1; }
    hipDeviceSynchronize();
    long * hosttrappower = (long *)calloc((int)(NUM_BLOCKS * THREADS_PER_BLOCK), sizeof(long));
    long * hosttrapsig_digs = (long *)calloc((int)(NUM_BLOCKS * THREADS_PER_BLOCK), sizeof(long));
    char * hosttrapdigits = (char *)calloc((int)(NUM_BLOCKS * THREADS_PER_BLOCK * max_digits), sizeof(char));
    long * hostmidpower = (long *)calloc((int)(NUM_BLOCKS * THREADS_PER_BLOCK), sizeof(long));
    long * hostmidsig_digs = (long *)calloc((int)(NUM_BLOCKS * THREADS_PER_BLOCK), sizeof(long));
    char * hostmiddigits = (char *)calloc((int)(NUM_BLOCKS * THREADS_PER_BLOCK * max_digits), sizeof(char));
    if ((hosttrappower == 0) || (hosttrapsig_digs == 0) || (hosttrapdigits == 0) ||
        (hostmidpower == 0) || (hostmidsig_digs == 0) || (hostmiddigits == 0)) {
        printf("\nError allocating memory on the CPU.\n"); 
        return 1; 
    }
    long * devicetrappower;
    long * devicetrapsig_digs;
    char * devicetrapdigits;
    long * devicemidpower;
    long * devicemidsig_digs;
    char * devicemiddigits;
    if (hipMalloc((void**)&devicetrappower, (int)(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(long))) 
        != hipSuccess) { printf("\nError allocating memory on GPU.\n"); return 1; }
    if (hipMalloc((void**)&devicetrapsig_digs, (int)(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(long))) 
        != hipSuccess) { printf("\nError allocating memory on GPU.\n"); return 1; }
    if (hipMalloc((void**)&devicetrapdigits, (int)(NUM_BLOCKS * THREADS_PER_BLOCK * max_digits * sizeof(char))) 
        != hipSuccess) { printf("\nError allocating memory on GPU.\n"); return 1; }
    if (hipMalloc((void**)&devicemidpower, (int)(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(long))) 
        != hipSuccess) { printf("\nError allocating memory on GPU.\n"); return 1; }
    if (hipMalloc((void**)&devicemidsig_digs, (int)(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(long))) 
        != hipSuccess) { printf("\nError allocating memory on GPU.\n"); return 1; }
    if (hipMalloc((void**)&devicemiddigits, (int)(NUM_BLOCKS * THREADS_PER_BLOCK * max_digits * sizeof(char))) 
        != hipSuccess) { printf("\nError allocating memory on GPU.\n"); return 1; }
    hipDeviceSynchronize();
    char * accepted_pi = "3.14159265358979323846264338327950288419716939937510"
        "58209749445923078164062862089986280348253421170679\0";
    char pi_printer[2];
    pi_printer[0] = '0';
    pi_printer[1] = '\0';
    
    // Split off worker threads. When dividing the work, if the number of 
    // threads does not evenly divide into the desired number of iterations,
    // give any extra iterations to the final thread. This gives the final
    // thread at most (num_threads - 1) extra iterations. Notice that this
    // is a 1D-grid of work, and we use function arguments this time. Also,
    // remember the number of threads is held constant, thanks to #defines, 
    // at NUM_BLOCKS * THREADS_PER_BLOCK. 
    dim3 numBlocks(NUM_BLOCKS);
    dim3 threadsPerBlock(THREADS_PER_BLOCK);
    calculate<<<numBlocks, threadsPerBlock>>>(devicetrappower, devicetrapsig_digs, 
        devicetrapdigits, devicemidpower, devicemidsig_digs, devicemiddigits, iterations, max_digits);
    hipDeviceSynchronize();
    
    // Copy results back from GPU
    if (hipMemcpy(hosttrappower, devicetrappower, (int)(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(long)), 
        hipMemcpyDeviceToHost) != hipSuccess) { printf("\nError copying memory from GPU.\n"); return 3; }
    if (hipMemcpy(hosttrapsig_digs, devicetrapsig_digs, (int)(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(long)), 
        hipMemcpyDeviceToHost) != hipSuccess) { printf("\nError copying memory from GPU.\n"); return 3; }
    if (hipMemcpy(hosttrapdigits, devicetrapdigits, (int)(NUM_BLOCKS * THREADS_PER_BLOCK * max_digits * sizeof(char)), 
        hipMemcpyDeviceToHost) != hipSuccess) { printf("\nError copying memory from GPU.\n"); return 3; }
    if (hipMemcpy(hostmidpower, devicemidpower, (int)(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(long)), 
        hipMemcpyDeviceToHost) != hipSuccess) { printf("\nError copying memory from GPU.\n"); return 3; }
    if (hipMemcpy(hostmidsig_digs, devicemidsig_digs, (int)(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(long)), 
        hipMemcpyDeviceToHost) != hipSuccess) { printf("\nError copying memory from GPU.\n"); return 3; }
    if (hipMemcpy(hostmiddigits, devicemiddigits, (int)(NUM_BLOCKS * THREADS_PER_BLOCK * max_digits * sizeof(char)), 
        hipMemcpyDeviceToHost) != hipSuccess) { printf("\nError copying memory from GPU.\n"); return 3; }
    hipDeviceSynchronize();
    if (hipFree(devicetrappower) != hipSuccess) { printf("\nError freeing GPU memory.\n"); return 3; }
    if (hipFree(devicetrapsig_digs) != hipSuccess) { printf("\nError freeing GPU memory.\n"); return 3; }
    if (hipFree(devicetrapdigits) != hipSuccess) { printf("\nError freeing GPU memory.\n"); return 3; }
    if (hipFree(devicemidpower) != hipSuccess) { printf("\nError freeing GPU memory.\n"); return 3; }
    if (hipFree(devicemidsig_digs) != hipSuccess) { printf("\nError freeing GPU memory.\n"); return 3; }
    if (hipFree(devicemiddigits) != hipSuccess) { printf("\nError freeing GPU memory.\n"); return 3; }
    
    // After worker threads end, clean up each of the partial sums
    bignum * trap = bignum_init(max_digits);
    bignum * mid = bignum_init(max_digits);
    bignum * temp = bignum_init(max_digits);
    bignum * simp = bignum_init(max_digits);
    if (trap == 0 || mid == 0 || temp == 0 || simp == 0) {
        printf("Error allocating memory. Now exiting.\n");
        return -1;
    }
    for (i = 0L; i < (NUM_BLOCKS * THREADS_PER_BLOCK); i++) {
        simp->power = hosttrappower[i];
        simp->sig_digs = hosttrapsig_digs[i];
        for (j = 0L; j < max_digits; j++) {
            simp->digits[(int)j] = hosttrapdigits[(int)((i * max_digits) + j)];
        }
        bignum_add(temp, trap, simp);
        bignum_reset(trap);
        bignum_reset(simp);
        bignum_set(trap, temp);
        bignum_reset(temp);
        simp->power = hostmidpower[i];
        simp->sig_digs = hostmidsig_digs[i];
        for (j = 0L; j < max_digits; j++) {
            simp->digits[(int)j] = hostmiddigits[(int)((i * max_digits) + j)];
        }
        bignum_add(temp, mid, simp);
        bignum_reset(mid);
        bignum_reset(simp);
        bignum_set(mid, temp);
        bignum_reset(temp);
    }

    // Finally, Simpson's Rule is applied
    bignum_mult_int(temp, mid, 2L);
    bignum_reset(mid);
    bignum_set(mid, temp);
    bignum_reset(temp);
    bignum_add(temp, trap, mid);
    bignum_reset(trap);
    bignum_set(trap, temp);
    bignum_reset(temp);
    bignum_divide_int(temp, trap, 3L);
    bignum_reset(trap);
    bignum_set(trap, temp);
    bignum_reset(temp);
    bignum_mult_int(simp, trap, 4L);
    long clock_end = (long)clock();
    printf("The calculated value of pi is ");
    bignum_print(simp, 0L);
    printf("\nThe actual value of pi is     3.");
    for (i = 0L; i < (max_digits - 1L); i++) { 
        // This may print an extra digit or two because, somewhere down in the
        // code, we're losing our last sig dig during normal math, but it's 
        // bubbling back up, and causing the final result to lose a place or
        // two. It's not a big deal, and I don't want to do anything about it, 
        // so we'll just have the ends of the numbers not line up. Whatever.
        pi_printer[0] = accepted_pi[(int)(i + 2L)];
        printf("%s", pi_printer);
    }
    printf("\nThe time taken to calculate this was %.2f seconds\n", 
        ((float)(clock_end - clock_start)) / (float)CLOCKS_PER_SEC);
        
    // Free global storage
    free(hosttrappower);
    free(hosttrapsig_digs);
    free(hosttrapdigits);
    free(hostmidpower);
    free(hostmidsig_digs);
    free(hostmiddigits);
    bignum_clear(trap);
    bignum_clear(mid);
    bignum_clear(simp);
    bignum_clear(temp);
    return 0;
}

// Function executed by each thread to incrementally calculate the overall value
__global__ void calculate (long * devicetrappower, long * devicetrapsig_digs, 
        char * devicetrapdigits, long * devicemidpower, long * devicemidsig_digs, 
        char * devicemiddigits, long iterations, long max_digits) 
{
    // Initialize needed variables and check for errors
    long threadid = threadIdx.x + (blockIdx.x * THREADS_PER_BLOCK);
    long lowlimit = threadid * (iterations / (NUM_BLOCKS * THREADS_PER_BLOCK));
    long highlimit = (((threadid + 1L) == (NUM_BLOCKS * THREADS_PER_BLOCK)) ? iterations : 
        ((threadid + 1L) * (iterations / (NUM_BLOCKS * THREADS_PER_BLOCK))));
    bignum * trap = bignum_init_gpu(max_digits);
    bignum * mid = bignum_init_gpu(max_digits);
    bignum * inverseiterations = bignum_init_gpu(max_digits);
    bignum * temp_holder = bignum_init_gpu(max_digits);
    bignum * temp_holder2 = bignum_init_gpu(max_digits);
    bignum * inc = bignum_init_gpu(max_digits);
    bignum * leftrect = bignum_init_gpu(max_digits);
    bignum * rightrect = bignum_init_gpu(max_digits);
    if (trap == 0 || mid == 0 || inverseiterations == 0 || temp_holder == 0 ||
        temp_holder2 == 0 || inc == 0 || leftrect == 0 || rightrect == 0) {
        return;
    }
    
    // Initialize values of needed variables
    bignum_set_int_gpu(temp_holder, iterations);
    bignum_int_divide_gpu(inverseiterations, 1L, temp_holder);
    bignum_reset_gpu(temp_holder);
    long i;
    long k = lowlimit;
    bignum_divide_int_gpu(temp_holder, inverseiterations, 2L);
    bignum_set_int_gpu(inc, k);
    bignum_mult_gpu(temp_holder2, inc, inverseiterations);
    bignum_reset_gpu(inc);
    bignum_set_gpu(inc, temp_holder2);
    bignum_reset_gpu(temp_holder2);
    bignum_add_gpu(temp_holder2, inc, temp_holder);
    bignum_reset_gpu(inc);
    bignum_set_gpu(inc, temp_holder2);
    bignum_reset_gpu(temp_holder2);
    bignum_reset_gpu(temp_holder);
    
    // Main iteration loop. Note that the values of inverseiterations, inc, 
    // mid, and trap are preserved across loop iterations, as is counter k.
    // inverseiterations is a constant that is stored for simplicity. Man, 
    // this is looking more and more like assembly...
    for (i = lowlimit; i < highlimit; i++) {
        // First, the trapezoid rule is used to estimate pi
        bignum_reset_gpu(leftrect);
        bignum_set_int_gpu(leftrect, k);
        bignum_mult_gpu(temp_holder2, leftrect, inverseiterations);
        bignum_reset_gpu(leftrect);
        bignum_set_gpu(leftrect, temp_holder2);
        bignum_reset_gpu(temp_holder2);
        k++;
        bignum_reset_gpu(rightrect);
        bignum_set_int_gpu(rightrect, k);
        bignum_mult_gpu(temp_holder2, rightrect, inverseiterations);
        bignum_reset_gpu(rightrect);
        bignum_set_gpu(rightrect, temp_holder2);
        bignum_reset_gpu(temp_holder2);
        bignum_add_gpu(temp_holder, leftrect, rightrect);
        bignum_divide_int_gpu(temp_holder2, temp_holder, 2L);
        bignum_reset_gpu(temp_holder);
        bignum_set_gpu(temp_holder, temp_holder2);
        bignum_reset_gpu(temp_holder2);
        bignum_mult_gpu(temp_holder2, temp_holder, temp_holder);
        bignum_reset_gpu(temp_holder);
        bignum_set_gpu(temp_holder, temp_holder2);
        bignum_reset_gpu(temp_holder2);
        bignum_add_int_gpu(temp_holder2, temp_holder, 1L);
        bignum_reset_gpu(temp_holder);
        bignum_set_gpu(temp_holder, temp_holder2);
        bignum_reset_gpu(temp_holder2);
        bignum_int_divide_gpu(temp_holder2, 1L, temp_holder);
        bignum_reset_gpu(temp_holder);
        bignum_set_gpu(temp_holder, temp_holder2);
        bignum_reset_gpu(temp_holder2);
        bignum_mult_gpu(temp_holder2, temp_holder, inverseiterations);
        bignum_reset_gpu(temp_holder);
        bignum_set_gpu(temp_holder, temp_holder2);
        bignum_reset_gpu(temp_holder2);
        bignum_add_gpu(temp_holder2, trap, temp_holder);
        bignum_reset_gpu(trap);
        bignum_set_gpu(trap, temp_holder2);
        bignum_reset_gpu(temp_holder2);
        bignum_reset_gpu(temp_holder);
        
        // Next, the midpoint rule is also used to estimate pi
        bignum_set_gpu(temp_holder, inc);
        bignum_add_gpu(temp_holder2, inc, inverseiterations);
        bignum_reset_gpu(inc);
        bignum_set_gpu(inc, temp_holder2);
        bignum_reset_gpu(temp_holder2);
        bignum_mult_gpu(temp_holder2, temp_holder, temp_holder);
        bignum_reset_gpu(temp_holder);
        bignum_set_gpu(temp_holder, temp_holder2);
        bignum_reset_gpu(temp_holder2);
        bignum_add_int_gpu(temp_holder2, temp_holder, 1L);
        bignum_reset_gpu(temp_holder);
        bignum_set_gpu(temp_holder, temp_holder2);
        bignum_reset_gpu(temp_holder2);
        bignum_int_divide_gpu(temp_holder2, 1L, temp_holder);
        bignum_reset_gpu(temp_holder);
        bignum_set_gpu(temp_holder, temp_holder2);
        bignum_reset_gpu(temp_holder2);
        bignum_mult_gpu(temp_holder2, temp_holder, inverseiterations);
        bignum_reset_gpu(temp_holder);
        bignum_set_gpu(temp_holder, temp_holder2);
        bignum_reset_gpu(temp_holder2);
        bignum_add_gpu(temp_holder2, mid, temp_holder);
        bignum_reset_gpu(mid);
        bignum_set_gpu(mid, temp_holder2);
        bignum_reset_gpu(temp_holder2);
        bignum_reset_gpu(temp_holder);
    }
    
    // Save partial result, clear memory, and exit
    devicetrappower[threadid] = trap->power;
    devicetrapsig_digs[threadid] = trap->sig_digs;
    for (i = 0; i < max_digits; i++) {
        devicetrapdigits[(threadid * max_digits) + i] = trap->digits[i];
    }
    devicemidpower[threadid] = mid->power;
    devicemidsig_digs[threadid] = mid->sig_digs;
    for (i = 0; i < max_digits; i++) {
        devicemiddigits[(threadid * max_digits) + i] = mid->digits[i];
    }
    bignum_clear_gpu(trap);
    bignum_clear_gpu(mid);
    bignum_clear_gpu(inverseiterations);
    bignum_clear_gpu(temp_holder);
    bignum_clear_gpu(temp_holder2);
    bignum_clear_gpu(inc);
    bignum_clear_gpu(leftrect);
    bignum_clear_gpu(rightrect);
}

// Create space for a bignum with the specified precision.
// Technically, it's also initialized if we interpret having zero
// significant digits as the number having a value of zero.
__host__ bignum * bignum_init(long int precision) {
    bignum * temp_ptr = (bignum *)calloc(1, sizeof(bignum));
    temp_ptr->digits = (char *)calloc((int)precision, sizeof(char));
    if ((temp_ptr->digits) == 0) { temp_ptr = 0; }
    temp_ptr->precision = precision;
    return temp_ptr;
}

// Resets a bignum's value to zero. memcpy isn't used because 
// why bring the string library into this just for this use?
__host__ void bignum_reset(bignum * numval) {
    if ((numval->sig_digs) > 0L) {
        long int i;
        for (i = 0L; i < numval->precision; i++) { numval->digits[(int)i] = '\0'; }
        numval->power = 0L;
        numval->sig_digs = 0L;
    }
    return;
}

// Free memory used by a bignum when we're done with it
__host__ void bignum_clear(bignum * oldnum) {
    free(oldnum->digits);
    free(oldnum);
    return;
}

// Set an instance of a bignum to an integer value. Note that if we can't 
// initialize the temp word we need for copying, we return false (value = 0). 
// We also assume that the number is non-negative since we only store 
// unsigned numbers. We assume the result is initialized/reset. Finally, 
// we handle zero specially by just resetting (again?) the result. Note that
// we explicitly assume the number to convert fits within the max number of
// digits. If we try to convert a number bigger than we can store, it won't work.
__host__ int bignum_set_int(bignum * numval, long int intval) {
    if (intval > 0L) {
        // Separate out the individual digits (stored backwards)
        char * temp_word = (char *)calloc((int)(numval->precision), sizeof(char));
        if (temp_word == 0) { return 0; }
        long int temp_int = intval;
        long int counter = 0L;
        while (temp_int > 0L) {
            temp_word[(int)counter] = (char)(temp_int % 10L);
            temp_int = temp_int / 10L;
            counter++;
        }
        
        // Detect any trailing zeros that we don't need to store
        numval->power = counter - 1L;
        long int leadingzeros = 0L;
        int hasleading = 1;
        while (hasleading == 1) {
            if (temp_word[(int)leadingzeros] != 0) { hasleading = 0; }
            else { leadingzeros++; }
        }
        
        // Store final result into actual bignum variable
        for (temp_int = 0L; temp_int < (counter - leadingzeros); temp_int++) {
            numval->digits[(int)temp_int] = temp_word[(int)(counter - temp_int - 1L)];
        }
        numval->sig_digs = counter - leadingzeros;
        free(temp_word);
        return 1;
    }
    else { bignum_reset(numval); return 1; }
}

// Set an instance of a bignum to the value of another bignum. We don't assume
// they're both the same precision; just use the precision of the new number.
// We do assume that the new number has already been initialized, though.
// strncpy is not used since it quits after seeing the first zero.
__host__ void bignum_set(bignum * newnum, bignum * oldnum) {
    if ((oldnum->sig_digs) > 0L) {
        newnum->power = oldnum->power;
        newnum->sig_digs = ((oldnum->sig_digs > newnum->precision) ? 
            (newnum->precision) : (oldnum->sig_digs));
        long int i;
        for (i = 0L; i < newnum->sig_digs; i++) {
            newnum->digits[(int)i] = oldnum->digits[(int)i];
        }
    }
    else { bignum_reset(newnum); }
    return;
}

// Use printf to print the number one digit at a time. There are a few cases:
// power > significant digits: pad end with zeros
// significant digits > power: fractional digit (non-integer)
// power is negative: total value less than 1
// The second argument is the maximum number of significant digits to print.
// If it's zero, then all available digits will be printed, maxing out at 
// the precision of the number (the total amount is could possibly store).
// Note that this is different from total digits printed: zeroes after a 
// decimal point but before the first significant digit don't count, and we
// make sure we print at least the integral part of the number (we only 
// chop off fractional portions).
__host__ void bignum_print(bignum * numval, long int maxdigits) {
    long int i;
    long int limit = numval->sig_digs;
    if (numval->sig_digs == 0L) { printf("0"); } else {
    if ((maxdigits > 0L) && (maxdigits < numval->sig_digs)) {
        limit = maxdigits;
    }
    if (numval->power < 0L) {
        printf("0.");
        for (i = 1L; i < (-1L * (numval->power)); i++) { printf("0"); }
        for (i = 0L; i < limit; i++) { 
            printf("%d", (int)(numval->digits[(int)i])); 
        }
    }
    else if (numval->sig_digs > (numval->power + 1L)) {
        for (i = 0L; i <= numval->power; i++) { 
            printf("%d", (int)(numval->digits[(int)i])); 
        }
        if (limit > (numval->power + 1L)) { printf("."); }
        for (i = (numval->power + 1L); i < limit; i++) { 
            printf("%d", (int)(numval->digits[(int)i])); 
        }
    }
    else { for (i = 0L; i < numval->sig_digs; i++) { 
        printf("%d", (int)(numval->digits[(int)i])); } 
    }
    if ((numval->power > 0L) && ((numval->power + 1L) > numval->sig_digs)) { 
        for (i = 0L; i < ((numval->power + 1L) - numval->sig_digs); i++) { 
            printf("0"); 
        } 
    } }
    fflush(stdout);
    return;
}

// Adds two bignums together and stores the result. Uses the functions to 
// reset and set the location of the result internally, so current contents of
// result operand will be overwritten. Like bignum_set_int, returns 1 if 
// addition was successful or 0 if an error occurred. A special shortcut is
// taken if either (or both) of the operands are zero. Note that it is possible 
// for large additions to cause underflow to zero. In that case, special care is
// taken to make sure the proper input operand is used. Note that we assume the
// precision of all three operands is the same. If it's not, something terrible
// like a seg fault or incorrect answer will probably occur. Most importantly, 
// the result operand CANNOT be the same as one of the input operands, since
// the result is clobbered immediately and used as a scratchpad. Note that this
// is also unsigned addition: not only does it not accept negative numbers, it
// also doesn't do subtraction (which, for that matter, isn't commutative).
__host__ int bignum_add(bignum * resultnum, bignum * leftnum, bignum * rightnum) {
    bignum_reset(resultnum);
    if ((leftnum->sig_digs == 0L) && (rightnum->sig_digs > 0L)) {
        bignum_set(resultnum, rightnum);
        return 1;
    }
    else if ((rightnum->sig_digs == 0L) && (leftnum->sig_digs > 0L)) {
        bignum_set(resultnum, leftnum);
        return 1;
    }
    else if ((leftnum->sig_digs == 0L) && (rightnum->sig_digs == 0L)) { return 1; }
    else {
        // First check for overshift:  if the larger number's power is too much
        // bigger than the smaller number's, the smaller will be completely lost,
        // and we'll just end up with the large number as the result.
        if ((((leftnum->power - rightnum->power) > 0) && 
            ((leftnum->power - rightnum->power) > resultnum->precision))) {
            bignum_set(resultnum, leftnum);
            return 1;
        }
        if ((((rightnum->power - leftnum->power) > 0) && 
            ((rightnum->power - leftnum->power) > resultnum->precision))) {
            bignum_set(resultnum, rightnum);
            return 1;
        }
        
        // Next, shift the smaller operand to match the larger one by copying
        // it into the result operand as a partial sum. Also copy over the 
        // power and total significant digits into the result.
        bignum * bigger;
        bignum * smaller;
        if ((leftnum->power - rightnum->power) >= 0L) {
            bigger = leftnum;
            smaller = rightnum;
        }
        else {
            bigger = rightnum;
            smaller = leftnum;
        }
        long int difference = bigger->power - smaller->power;
        long int startdigit = smaller->sig_digs + difference;
        long int transfertotal = smaller->sig_digs;
        if (startdigit > resultnum->precision) {
            startdigit = resultnum->precision - difference;
            transfertotal = startdigit;
        }
        long int startdigitcopy = startdigit;
        startdigit--;
        long int i;
        for (i = 0L; i < transfertotal; i++) {
            if ((startdigit - difference) >= 0L) {
                resultnum->digits[(int)startdigit] = 
                    smaller->digits[(int)(startdigit - difference)];
            }
            startdigit--;
        }
        
        // Now the main addition loop: loop through each digit and add it.
        // The carry from the previous digit will add to the current one.
        // Note that we detect any trailing zeros to take from the sig_digs.
        // Also, copy over the power and significant digits
        resultnum->power = bigger->power;
        resultnum->sig_digs = startdigitcopy;
        if (bigger->sig_digs > resultnum->sig_digs) {
            resultnum->sig_digs = bigger->sig_digs;
            startdigitcopy = resultnum->sig_digs;
        }
        int trailingzeros = 1;
        long int zerocount = 0L;
        char carry = 0;
        for (i = 0L; i < resultnum->sig_digs; i++) {
            resultnum->digits[(int)(startdigitcopy - i - 1L)] += 
                (bigger->digits[(int)(startdigitcopy - i - 1L)] + carry);
            if (resultnum->digits[(int)(startdigitcopy - i - 1L)] >= 10) {
                resultnum->digits[(int)(startdigitcopy - i - 1L)] -= 10;
                carry = 1;
            } else { carry = 0; }
            if (trailingzeros == 1) {
                if (resultnum->digits[(int)(startdigitcopy - i - 1L)] == '\0') {
                    zerocount++;
                } else { trailingzeros = 0; }
            }
        }
        
        // If we've got trailing zeros, subtract them from the final count of
        // sig_digs. Also, if we have a carry, we need to shift everything...
        resultnum->sig_digs -= zerocount;
        if (carry > 0) {
            transfertotal = resultnum->sig_digs;
            if (transfertotal == resultnum->precision) { transfertotal--; }
            startdigitcopy = transfertotal - 1L;
            for (i = 0L; i < transfertotal; i++) {
                if (startdigitcopy >= 0L) {
                    resultnum->digits[(int)(startdigitcopy + 1L)] =
                        resultnum->digits[(int)startdigitcopy];
                }
                else if ((startdigitcopy + 1L) >= 0L) {
                    resultnum->digits[(int)(startdigitcopy + 1L)] = 0;
                }
                startdigitcopy--;
            }
            resultnum->digits[0] = carry;
            resultnum->power++;
            resultnum->sig_digs++;
        }
        if (resultnum->sig_digs > resultnum->precision) {
            resultnum->sig_digs = resultnum->precision;
        }
        return 1;
    }
}

// A convenience wrapper that temporarily creates a new bignum out of the 
// given integer, calls bignum_add with it and the other operand, and deletes
// the temporary bignum before exiting. Any problems that bignum_add encounters
// are passed back up through this function and returned to the caller.
__host__ int bignum_add_int(bignum * resultnum, bignum * leftnum, long int rightint) {
    bignum_reset(resultnum);
    if ((rightint == 0L) && (leftnum->sig_digs > 0L)) {
        bignum_set(resultnum, leftnum);
        return 1;
    }
    else if ((leftnum->sig_digs == 0L) && (rightint > 0L)) {
        return bignum_set_int(resultnum, rightint);
    }
    else if ((leftnum->sig_digs == 0L) && (rightint == 0L)) { return 1; }
    else {
        bignum * tempnum = bignum_init(resultnum->precision);
        if (tempnum == 0) { return 0; }
        if (bignum_set_int(tempnum, rightint) == 0) {
            bignum_clear(tempnum);
            return 0;
        }
        int retval = bignum_add(resultnum, leftnum, tempnum);
        bignum_clear(tempnum);
        return retval;
    }
}

// Multiplies two bignums together and stores the result. Like add, uses 
// functions to reset and set the location of the result, and returns 1 upon
// success or 0 if an error occurred. A special shortcut is taken if either
// operand is zero, since the result will thus also be zero. Note that we assume
// the precision of all three operands is the same. If it's not, something 
// terrible like a seg fault or incorrect answer will probably occur. Most 
// importantly, the result operand CANNOT be the same as one of the input
// operands, since the result is clobbered immediately and used as a scratchpad.
// Also, note that this is unsigned: it assumes both operands are positive.
__host__ int bignum_mult(bignum * resultnum, bignum * leftnum, bignum * rightnum) {
    bignum_reset(resultnum);
    if ((leftnum->sig_digs == 0L) || (rightnum->sig_digs == 0L)) { return 1; }
    else {
        // Initialize the scratchpad and find the digit limits
        char * temp_word = (char *)calloc((int)(2L * (resultnum->precision)), sizeof(char));
        if (temp_word == 0) { return 0; }
        bignum * bigger;
        bignum * smaller;
        if (((signed long int)leftnum->sig_digs - (signed long int)rightnum->sig_digs) >= 0L) {
            bigger = leftnum;
            smaller = rightnum;
        }
        else if ((rightnum->sig_digs - leftnum->sig_digs) > 0L) {
            bigger = rightnum;
            smaller = leftnum;
        }
        long int bigstart = (bigger->sig_digs) - 1L;
        long int smallstart = (smaller->sig_digs) - 1L;
        long int bigcounter, smallcounter;
        char carry = 0;
        
        // Perform the shift-addition loop. We choose to loop over each
        // digit of the smaller number for fewer overall iterations. If
        // the current bigloop has a zero, we can just skip that iteration.
        // Also, record the final carry, power, and sig_digs values. 
        for (bigcounter = 0L; bigcounter < (smaller->sig_digs); bigcounter++) {
            if (smaller->digits[(int)(smallstart - bigcounter)] != '\0') {
                carry = 0;
                for(smallcounter = 0L; smallcounter < (bigger->sig_digs); smallcounter++) {
                    temp_word[(int)((2L * (resultnum->precision)) - smallcounter - 
                        bigcounter - 1L)] += (carry + (smaller->digits[(int)(smallstart - 
                        bigcounter)] * bigger->digits[(int)(bigstart - smallcounter)]));
                    carry = temp_word[(int)((2L * (resultnum->precision)) - 
                        smallcounter - bigcounter - 1L)] / 10;
                    temp_word[(int)((2L * (resultnum->precision)) - smallcounter - 
                        bigcounter - 1L)] %= 10;
                }
                temp_word[(int)((2L * (resultnum->precision)) - bigcounter -
                    (bigger->sig_digs) - 1L)] = carry;
            }
        }
        resultnum->power = ((bigger->power) + (smaller->power));
        resultnum->sig_digs = ((bigger->sig_digs) + (smaller->sig_digs));
        
        // Adjust for lack of a final carry or trailing zeros.
        if (carry < 1) { 
            (resultnum->sig_digs)--; 
            (resultnum->power)--; 
        }
        (resultnum->power)++; 
        int trailingzeros = 1;
        long int zerocount = 0L;
        long int i = (2L * (resultnum->precision) - 1L); 
        while (trailingzeros == 1) {
            if (temp_word[(int)i] == '\0') {
                zerocount++;
            } else { trailingzeros = 0; }
            i--;
        }
        resultnum->sig_digs -= zerocount;
        if ((resultnum->sig_digs) > (resultnum->precision)) {
            resultnum->sig_digs = (resultnum->precision);
        }
        
        // Finally, copy from the temp word into the result, taking into 
        // account any digits we may lose due to precision.
        long int tempstart = (2L * (resultnum->precision)) - ((bigger->sig_digs) + 
            (smaller->sig_digs));
        if (carry < 1) { tempstart++; }
        for (i = 0L; i < (resultnum->sig_digs); i++) {
            resultnum->digits[(int)i] = temp_word[(int)(tempstart + i)];
        }
        free(temp_word);
        return 1;
    }
}

// Like bignum_add_int, a convenience wrapper that creates a temporary bignum
// out of the integer and passes it to bignum_mult. Any problems encountered 
// in client functions are passed back up to the original caller.
__host__ int bignum_mult_int(bignum * resultnum, bignum * leftnum, long int rightint) {
    bignum_reset(resultnum);
    if ((leftnum->sig_digs == 0L) || (rightint == 0L)) { return 1; }
    else {
        bignum * tempnum = bignum_init(resultnum->precision);
        if (tempnum == 0) { return 0; }
        if (bignum_set_int(tempnum, rightint) == 0) {
            bignum_clear(tempnum);
            return 0;
        }
        int retval = bignum_mult(resultnum, leftnum, tempnum);
        bignum_clear(tempnum);
        return retval;
    }
}

// Divides two bignums. Taken in terms of a fraction, leftnum is the numerator 
// and rightnum is the denominator. Performs an explicit check to make sure
// the denominator is not zero, and returns 0 (an error) if it is. Returns 1 upon 
// success or 0 if an error occurs. A special shortcut is taken if the numerator is 
// zero. Note that we assume the precision of all three operands is the same. If it's 
// not, something terrible like a seg fault or incorrect answer will probably occur. 
// Most importantly, the result operand CANNOT be the same as one of the input 
// operands, since the result is clobbered immediately and used as a scratchpad.
// Also, note that this is unsigned: it assumes both operands are positive.
__host__ int bignum_divide(bignum * resultnum, bignum * numerator, bignum * denominator) {
    bignum_reset(resultnum);
    if (denominator->sig_digs == 0L) { return 0; }
    else if (numerator->sig_digs == 0L) { return 1; }
    else {
        // Initialize the scratchpad and initially copy the numerator into it.
        // Also initialize the result's power.
        char * temp_word = (char *)calloc((int)(2L * 
            (resultnum->precision) + 2L), sizeof(char)); // May only need to be + 1L
        if (temp_word == 0) { return 0; }
        long int i;
        for (i = 0L; i < numerator->sig_digs; i++) {
            temp_word[(int)(i + 1L)] = numerator->digits[(int)i];
        }
        resultnum->power = (numerator->power - denominator->power);
        long int sigdigctr = 0L;
        long int numeratorindex = 0L;
        
        // First see if we need to "shift" the numerator by comparing it.
        i = ((denominator->sig_digs) - 1L);
        int denom_bigger = 1;
        while ((i >= 0L) && (denom_bigger == 1)) {
            if ((denominator->digits[(int)((denominator->sig_digs) - i - 1L)]) > 
                (temp_word[(int)((denominator->sig_digs) - i)])) {
                i = 0L;
            }
            else if ((denominator->digits[(int)((denominator->sig_digs) - 
                i - 1L)]) < (temp_word[(int)((denominator->sig_digs) - i)])) {
                denom_bigger = 0;
            }
            else if (((denominator->digits[(int)((denominator->sig_digs) - i - 
                1L)]) == (temp_word[(int)((denominator->sig_digs) - i)])) && (i == 0L)) {
                denom_bigger = 0;
            }
            i--;
        }
        if (denom_bigger == 1) { 
            numeratorindex++; 
            (resultnum->power)--;
        }
        
        // Now the main division loop. Note that there's two ways to terminate:
        // either we've filled the entire precision of the result word and are
        // forced to truncate our result, or our answer divides exactly. In the
        // second case, once we've exhausted the numerator's significant digits
        // and our temp word contains nothing but zeros, we can end early since
        // all subsequent iterations would contribute only zeros as well. Note
        // that special care will be taken to detect extra zeros at the end of
        // the result so that the sig_digs is recorded correctly. Also, we don't
        // round, we truncate, which doesn't minimize error.
        int nonzero = 1;
        while ((sigdigctr < (resultnum->precision)) && (nonzero == 1)) {
            // First run the subtraction loop.
            char current_digit = 0;
            int numer_bigger = 1;
            while (numer_bigger == 1) {
                // To subtract, first run a comparison to see if the numerator
                // is bigger. If it is, increment the counter and subtract. 
                i = ((denominator->sig_digs) - 1L);
                denom_bigger = 1;
                if (temp_word[(int)numeratorindex] > 0) { denom_bigger = 0; }
                while ((i >= 0L) && (denom_bigger == 1)) {
                    if ((denominator->digits[(int)((denominator->sig_digs) - 
                        i - 1L)]) > (temp_word[(int)((denominator->sig_digs) + 
                        numeratorindex - i)])) {
                        i = 0L;
                    }
                    else if ((denominator->digits[(int)((denominator->sig_digs) - 
                        i - 1L)]) < (temp_word[(int)((denominator->sig_digs) + 
                        numeratorindex - i)])) {
                        denom_bigger = 0;
                    }
                    else if (((denominator->digits[(int)((denominator->sig_digs) - 
                        i - 1L)]) == (temp_word[(int)((denominator->sig_digs) + 
                        numeratorindex - i)])) && (i == 0L)) {
                        denom_bigger = 0;
                    }
                    i--;
                }
                if (denom_bigger == 1) { 
                    numer_bigger = 0;
                }
                
                // Increment counter and perform subtraction loop.
                if (numer_bigger == 1) {
                    current_digit++;
                    for (i = 0L; i < (denominator->sig_digs); i++) {
                        temp_word[(int)((denominator->sig_digs) + 
                            numeratorindex - i)] -= (denominator->digits[
                            (int)((denominator->sig_digs) - i - 1L)]);
                        if ((temp_word[(int)((denominator->sig_digs) + 
                            numeratorindex - i)]) < 0) {
                            temp_word[(int)((denominator->sig_digs) + 
                                numeratorindex - i)] += 10L;
                            (temp_word[(int)((denominator->sig_digs) + 
                                numeratorindex - i - 1L)]) -= 1L;
                        }
                    }
                }
            }
            
            // If we're past all of the numerator's significant digits, run
            // zero detection on it to see if we can end early.
            if (sigdigctr > (numerator->sig_digs)) { // May only need to be >=
                long int zerocounter = 0L; 
                i = 0L;
                while ((i == zerocounter) && (i <= (denominator->sig_digs))) {
                    if ((temp_word[(int)(numeratorindex + i)]) < 1) { zerocounter++; }
                    i++;
                }
                if (zerocounter == ((denominator->sig_digs) + 1L)) { nonzero = 0; }
            }
            
            // Once we have obtained the proper digit in the result, save it.
            if (sigdigctr < resultnum->precision) {
                resultnum->digits[(int)sigdigctr] = current_digit;
            }
            sigdigctr++;
            numeratorindex++;
        }
        
        // Record the result's sig digs, taking care to detect trailing zeros.
        resultnum->sig_digs = sigdigctr;
        int trailingzeros = 1;
        long int zerocount = 0L;
        i = sigdigctr - 1L; 
        while (trailingzeros == 1) {
            if (resultnum->digits[(int)i] == '\0') {
                zerocount++;
            } else { trailingzeros = 0; }
            i--;
        }
        (resultnum->sig_digs) -= zerocount;
        free (temp_word);
        return 1;
    }
}

// A convenience wrapper that creates a temporary bignum out of the integer. 
// Since division is not commutative, two wrappers are given. Any problems 
// encountered in client functions are passed back up to the original caller.
__host__ int bignum_int_divide(bignum * resultnum, long int leftint, bignum * rightnum) {
    bignum_reset(resultnum);
    if (rightnum->sig_digs == 0L) { return 0; }
    else if (leftint == 0L) { return 1; }
    else {
        bignum * tempnum = bignum_init(resultnum->precision);
        if (tempnum == 0) { return 0; }
        if (bignum_set_int(tempnum, leftint) == 0) {
            bignum_clear(tempnum);
            return 0;
        }
        int retval = bignum_divide(resultnum, tempnum, rightnum);
        bignum_clear(tempnum);
        return retval;
    }
}

// A convenience wrapper that creates a temporary bignum out of the integer. 
// Since division is not commutative, two wrappers are given. Any problems 
// encountered in client functions are passed back up to the original caller.
__host__ int bignum_divide_int(bignum * resultnum, bignum * leftnum, long int rightint) {
    bignum_reset(resultnum);
    if (rightint == 0L) { return 0; }
    else if (leftnum->sig_digs == 0L) { return 1; }
    else {
        bignum * tempnum = bignum_init(resultnum->precision);
        if (tempnum == 0) { return 0; }
        if (bignum_set_int(tempnum, rightint) == 0) {
            bignum_clear(tempnum);
            return 0;
        }
        int retval = bignum_divide(resultnum, leftnum, tempnum);
        bignum_clear(tempnum);
        return retval;
    }
}

// Create space for a bignum with the specified precision.
// Technically, it's also initialized if we interpret having zero
// significant digits as the number having a value of zero.
__device__ bignum * bignum_init_gpu(long int precision) {
    bignum * temp_ptr = (bignum *)malloc(sizeof(bignum));
    if (temp_ptr == 0) { return temp_ptr; }
    temp_ptr->digits = (char *)malloc((int)(precision * sizeof(char)));
    if ((temp_ptr->digits) == 0) { temp_ptr = 0; return temp_ptr; }
    int i;
    for (i = 0; i < precision; i++) { temp_ptr->digits[i] = '\0'; }
    temp_ptr->power = 0L;
    temp_ptr->sig_digs = 0L;
    temp_ptr->precision = precision;
    return temp_ptr;
}

// Resets a bignum's value to zero. memcpy isn't used because 
// why bring the string library into this just for this use?
__device__ void bignum_reset_gpu(bignum * numval) {
    if ((numval->sig_digs) > 0L) {
        long int i;
        for (i = 0L; i < numval->precision; i++) { numval->digits[(int)i] = '\0'; }
        numval->power = 0L;
        numval->sig_digs = 0L;
    }
    return;
}

// Free memory used by a bignum when we're done with it
__device__ void bignum_clear_gpu(bignum * oldnum) {
    free(oldnum->digits);
    free(oldnum);
    return;
}

// Set an instance of a bignum to an integer value. Note that if we can't 
// initialize the temp word we need for copying, we return false (value = 0). 
// We also assume that the number is non-negative since we only store 
// unsigned numbers. We assume the result is initialized/reset. Finally, 
// we handle zero specially by just resetting (again?) the result. Note that
// we explicitly assume the number to convert fits within the max number of
// digits. If we try to convert a number bigger than we can store, it won't work.
__device__ int bignum_set_int_gpu(bignum * numval, long int intval) {
    if (intval > 0L) {
        
        // Separate out the individual digits (stored backwards)
        char * temp_word = (char *)malloc((int)(numval->precision * sizeof(char)));
        if (temp_word == 0) { return 0; }
        long int i;
        for (i = 0; i < numval->precision; i++) { temp_word[(int)i] = '\0'; }
        long int temp_int = intval;
        long int counter = 0L;
        while (temp_int > 0L) {
            temp_word[(int)counter] = (char)(temp_int % 10L);
            temp_int = temp_int / 10L;
            counter++;
        }
        
        // Detect any trailing zeros that we don't need to store
        numval->power = counter - 1L;
        long int leadingzeros = 0L;
        int hasleading = 1;
        while (hasleading == 1) {
            if (temp_word[(int)leadingzeros] != 0) { hasleading = 0; }
            else { leadingzeros++; }
        }
        
        // Store final result into actual bignum variable
        for (temp_int = 0L; temp_int < (counter - leadingzeros); temp_int++) {
            numval->digits[(int)temp_int] = temp_word[(int)(counter - temp_int - 1L)];
        }
        numval->sig_digs = counter - leadingzeros;
        free(temp_word);
        return 1;
    }
    else { bignum_reset_gpu(numval); return 1; }
}

// Set an instance of a bignum to the value of another bignum. We don't assume
// they're both the same precision; just use the precision of the new number.
// We do assume that the new number has already been initialized, though.
// strncpy is not used since it quits after seeing the first zero.
__device__ void bignum_set_gpu(bignum * newnum, bignum * oldnum) {
    if ((oldnum->sig_digs) > 0L) {
        newnum->power = oldnum->power;
        newnum->sig_digs = ((oldnum->sig_digs > newnum->precision) ? 
            (newnum->precision) : (oldnum->sig_digs));
        long int i;
        for (i = 0L; i < newnum->sig_digs; i++) {
            newnum->digits[(int)i] = oldnum->digits[(int)i];
        }
    }
    else { bignum_reset_gpu(newnum); }
    return;
}

// Adds two bignums together and stores the result. Uses the functions to 
// reset and set the location of the result internally, so current contents of
// result operand will be overwritten. Like bignum_set_int, returns 1 if 
// addition was successful or 0 if an error occurred. A special shortcut is
// taken if either (or both) of the operands are zero. Note that it is possible 
// for large additions to cause underflow to zero. In that case, special care is
// taken to make sure the proper input operand is used. Note that we assume the
// precision of all three operands is the same. If it's not, something terrible
// like a seg fault or incorrect answer will probably occur. Most importantly, 
// the result operand CANNOT be the same as one of the input operands, since
// the result is clobbered immediately and used as a scratchpad. Note that this
// is also unsigned addition: not only does it not accept negative numbers, it
// also doesn't do subtraction (which, for that matter, isn't commutative).
__device__ int bignum_add_gpu(bignum * resultnum, bignum * leftnum, bignum * rightnum) {
    bignum_reset_gpu(resultnum);
    if ((leftnum->sig_digs == 0L) && (rightnum->sig_digs > 0L)) {
        bignum_set_gpu(resultnum, rightnum);
        return 1;
    }
    else if ((rightnum->sig_digs == 0L) && (leftnum->sig_digs > 0L)) {
        bignum_set_gpu(resultnum, leftnum);
        return 1;
    }
    else if ((leftnum->sig_digs == 0L) && (rightnum->sig_digs == 0L)) { return 1; }
    else {
        
        // First check for overshift:  if the larger number's power is too much
        // bigger than the smaller number's, the smaller will be completely lost,
        // and we'll just end up with the large number as the result.
        if ((((leftnum->power - rightnum->power) > 0) && 
            ((leftnum->power - rightnum->power) > resultnum->precision))) {
            bignum_set_gpu(resultnum, leftnum);
            return 1;
        }
        if ((((rightnum->power - leftnum->power) > 0) && 
            ((rightnum->power - leftnum->power) > resultnum->precision))) {
            bignum_set_gpu(resultnum, rightnum);
            return 1;
        }
        
        // Next, shift the smaller operand to match the larger one by copying
        // it into the result operand as a partial sum. Also copy over the 
        // power and total significant digits into the result.
        bignum * bigger;
        bignum * smaller;
        if ((leftnum->power - rightnum->power) >= 0L) {
            bigger = leftnum;
            smaller = rightnum;
        }
        else {
            bigger = rightnum;
            smaller = leftnum;
        }
        long int difference = bigger->power - smaller->power;
        long int startdigit = smaller->sig_digs + difference;
        long int transfertotal = smaller->sig_digs;
        if (startdigit > resultnum->precision) {
            startdigit = resultnum->precision - difference;
            transfertotal = startdigit;
        }
        long int startdigitcopy = startdigit;
        startdigit--;
        long int i;
        for (i = 0L; i < transfertotal; i++) {
            if ((startdigit - difference) >= 0L) {
                resultnum->digits[(int)startdigit] = 
                    smaller->digits[(int)(startdigit - difference)];
            }
            startdigit--;
        }
        
        // Now the main addition loop: loop through each digit and add it.
        // The carry from the previous digit will add to the current one.
        // Note that we detect any trailing zeros to take from the sig_digs.
        // Also, copy over the power and significant digits
        resultnum->power = bigger->power;
        resultnum->sig_digs = startdigitcopy;
        if (bigger->sig_digs > resultnum->sig_digs) {
            resultnum->sig_digs = bigger->sig_digs;
            startdigitcopy = resultnum->sig_digs;
        }
        int trailingzeros = 1;
        long int zerocount = 0L;
        char carry = 0;
        for (i = 0L; i < resultnum->sig_digs; i++) {
            resultnum->digits[(int)(startdigitcopy - i - 1L)] += 
                (bigger->digits[(int)(startdigitcopy - i - 1L)] + carry);
            if (resultnum->digits[(int)(startdigitcopy - i - 1L)] >= 10) {
                resultnum->digits[(int)(startdigitcopy - i - 1L)] -= 10;
                carry = 1;
            } else { carry = 0; }
            if (trailingzeros == 1) {
                if (resultnum->digits[(int)(startdigitcopy - i - 1L)] == '\0') {
                    zerocount++;
                } else { trailingzeros = 0; }
            }
        }
        
        // If we've got trailing zeros, subtract them from the final count of
        // sig_digs. Also, if we have a carry, we need to shift everything...
        resultnum->sig_digs -= zerocount;
        if (carry > 0) {
            transfertotal = resultnum->sig_digs;
            if (transfertotal == resultnum->precision) { transfertotal--; }
            startdigitcopy = transfertotal - 1L;
            for (i = 0L; i < transfertotal; i++) {
                if (startdigitcopy >= 0L) {
                    resultnum->digits[(int)(startdigitcopy + 1L)] =
                        resultnum->digits[(int)startdigitcopy];
                }
                else if ((startdigitcopy + 1L) >= 0L) {
                    resultnum->digits[(int)(startdigitcopy + 1L)] = '\0';
                }
                startdigitcopy--;
            }
            resultnum->digits[0] = carry;
            resultnum->power++;
            resultnum->sig_digs++;
        }
        if (resultnum->sig_digs > resultnum->precision) {
            resultnum->sig_digs = resultnum->precision;
        }
        return 1;
    }
}

// A convenience wrapper that temporarily creates a new bignum out of the 
// given integer, calls bignum_add with it and the other operand, and deletes
// the temporary bignum before exiting. Any problems that bignum_add encounters
// are passed back up through this function and returned to the caller.
__device__ int bignum_add_int_gpu(bignum * resultnum, bignum * leftnum, long int rightint) {
    bignum_reset_gpu(resultnum);
    if ((rightint == 0L) && (leftnum->sig_digs > 0L)) {
        bignum_set_gpu(resultnum, leftnum);
        return 1;
    }
    else if ((leftnum->sig_digs == 0L) && (rightint > 0L)) {
        return bignum_set_int_gpu(resultnum, rightint);
    }
    else if ((leftnum->sig_digs == 0L) && (rightint == 0L)) { return 1; }
    else {
        bignum * tempnum = bignum_init_gpu(resultnum->precision);
        if (tempnum == 0) { return 0; }
        if (bignum_set_int_gpu(tempnum, rightint) == 0) {
            bignum_clear_gpu(tempnum);
            return 0;
        }
        int retval = bignum_add_gpu(resultnum, leftnum, tempnum);
        bignum_clear_gpu(tempnum);
        return retval;
    }
}

// Multiplies two bignums together and stores the result. Like add, uses 
// functions to reset and set the location of the result, and returns 1 upon
// success or 0 if an error occurred. A special shortcut is taken if either
// operand is zero, since the result will thus also be zero. Note that we assume
// the precision of all three operands is the same. If it's not, something 
// terrible like a seg fault or incorrect answer will probably occur. Most 
// importantly, the result operand CANNOT be the same as one of the input
// operands, since the result is clobbered immediately and used as a scratchpad.
// Also, note that this is unsigned: it assumes both operands are positive.
__device__ int bignum_mult_gpu(bignum * resultnum, bignum * leftnum, bignum * rightnum) {
    bignum_reset_gpu(resultnum);
    if ((leftnum->sig_digs == 0L) || (rightnum->sig_digs == 0L)) { return 1; }
    else {
        
        // Initialize the scratchpad and find the digit limits
        char * temp_word = (char *)malloc((int)(2L * (resultnum->precision) * sizeof(char)));
        if (temp_word == 0) { return 0; }
        long int i;
        for (i = 0; i < (2L * resultnum->precision); i++) { temp_word[(int)i] = '\0'; }
        bignum * bigger;
        bignum * smaller;
        if (((signed long int)leftnum->sig_digs - (signed long int)rightnum->sig_digs) >= 0L) {
            bigger = leftnum;
            smaller = rightnum;
        }
        else if ((rightnum->sig_digs - leftnum->sig_digs) > 0L) {
            bigger = rightnum;
            smaller = leftnum;
        }
        long int bigstart = (bigger->sig_digs) - 1L;
        long int smallstart = (smaller->sig_digs) - 1L;
        long int bigcounter, smallcounter;
        char carry = 0;
        
        // Perform the shift-addition loop. We choose to loop over each
        // digit of the smaller number for fewer overall iterations. If
        // the current bigloop has a zero, we can just skip that iteration.
        // Also, record the final carry, power, and sig_digs values. 
        for (bigcounter = 0L; bigcounter < (smaller->sig_digs); bigcounter++) {
            if (smaller->digits[(int)(smallstart - bigcounter)] != '\0') {
                carry = 0;
                for(smallcounter = 0L; smallcounter < (bigger->sig_digs); smallcounter++) {
                    temp_word[(int)((2L * (resultnum->precision)) - smallcounter - 
                        bigcounter - 1L)] += (carry + (smaller->digits[(int)(smallstart - 
                        bigcounter)] * bigger->digits[(int)(bigstart - smallcounter)]));
                    carry = temp_word[(int)((2L * (resultnum->precision)) - 
                        smallcounter - bigcounter - 1L)] / 10;
                    temp_word[(int)((2L * (resultnum->precision)) - smallcounter - 
                        bigcounter - 1L)] %= 10;
                }
                temp_word[(int)((2L * (resultnum->precision)) - bigcounter -
                    (bigger->sig_digs) - 1L)] = carry;
            }
        }
        resultnum->power = ((bigger->power) + (smaller->power));
        resultnum->sig_digs = ((bigger->sig_digs) + (smaller->sig_digs));
        
        // Adjust for lack of a final carry or trailing zeros.
        if (carry < 1) { 
            (resultnum->sig_digs)--; 
            (resultnum->power)--; 
        }
        (resultnum->power)++; 
        int trailingzeros = 1;
        long int zerocount = 0L;
        i = (2L * (resultnum->precision) - 1L); 
        while (trailingzeros == 1) {
            if (temp_word[(int)i] == '\0') {
                zerocount++;
            } else { trailingzeros = 0; }
            i--;
        }
        resultnum->sig_digs -= zerocount;
        if ((resultnum->sig_digs) > (resultnum->precision)) {
            resultnum->sig_digs = (resultnum->precision);
        }
        
        // Finally, copy from the temp word into the result, taking into 
        // account any digits we may lose due to precision.
        long int tempstart = (2L * (resultnum->precision)) - ((bigger->sig_digs) + 
            (smaller->sig_digs));
        if (carry < 1) { tempstart++; }
        for (i = 0L; i < (resultnum->sig_digs); i++) {
            resultnum->digits[(int)i] = temp_word[(int)(tempstart + i)];
        }
        free(temp_word);
        return 1;
    }
}

// Like bignum_add_int, a convenience wrapper that creates a temporary bignum
// out of the integer and passes it to bignum_mult. Any problems encountered 
// in client functions are passed back up to the original caller.
__device__ int bignum_mult_int_gpu(bignum * resultnum, bignum * leftnum, long int rightint) {
    bignum_reset_gpu(resultnum);
    if ((leftnum->sig_digs == 0L) || (rightint == 0L)) { return 1; }
    else {
        bignum * tempnum = bignum_init_gpu(resultnum->precision);
        if (tempnum == 0) { return 0; }
        if (bignum_set_int_gpu(tempnum, rightint) == 0) {
            bignum_clear_gpu(tempnum);
            return 0;
        }
        int retval = bignum_mult_gpu(resultnum, leftnum, tempnum);
        bignum_clear_gpu(tempnum);
        return retval;
    }
}

// Divides two bignums. Taken in terms of a fraction, leftnum is the numerator 
// and rightnum is the denominator. Performs an explicit check to make sure
// the denominator is not zero, and returns 0 (an error) if it is. Returns 1 upon 
// success or 0 if an error occurs. A special shortcut is taken if the numerator is 
// zero. Note that we assume the precision of all three operands is the same. If it's 
// not, something terrible like a seg fault or incorrect answer will probably occur. 
// Most importantly, the result operand CANNOT be the same as one of the input 
// operands, since the result is clobbered immediately and used as a scratchpad.
// Also, note that this is unsigned: it assumes both operands are positive.
__device__ int bignum_divide_gpu(bignum * resultnum, bignum * numerator, bignum * denominator) {
    bignum_reset_gpu(resultnum);
    if (denominator->sig_digs == 0L) { return 0; }
    else if (numerator->sig_digs == 0L) { return 1; }
    else {
        
        // Initialize the scratchpad and initially copy the numerator into it.
        // Also initialize the result's power.
        char * temp_word = (char *)malloc((int)(2L * 
            (resultnum->precision) + 2L * sizeof(char))); // May only need to be + 1L
        if (temp_word == 0) { return 0; }
        long int i;
        temp_word[0] = '\0';
        for (i = 0L; i < numerator->sig_digs; i++) {
            temp_word[(int)(i + 1L)] = numerator->digits[(int)i];
        }
        for (i = (1L + numerator->sig_digs); i < 
            (2L * resultnum->precision + 2L); i++) { temp_word[(int)i] = '\0'; }
        
        resultnum->power = (numerator->power - denominator->power);
        long int sigdigctr = 0L;
        long int numeratorindex = 0L;
        
        // First see if we need to "shift" the numerator by comparing it.
        i = ((denominator->sig_digs) - 1L);
        int denom_bigger = 1;
        while ((i >= 0L) && (denom_bigger == 1)) {
            if ((denominator->digits[(int)((denominator->sig_digs) - i - 1L)]) > 
                (temp_word[(int)((denominator->sig_digs) - i)])) {
                i = 0L;
            }
            else if ((denominator->digits[(int)((denominator->sig_digs) - 
                i - 1L)]) < (temp_word[(int)((denominator->sig_digs) - i)])) {
                denom_bigger = 0;
            }
            else if (((denominator->digits[(int)((denominator->sig_digs) - i - 
                1L)]) == (temp_word[(int)((denominator->sig_digs) - i)])) && (i == 0L)) {
                denom_bigger = 0;
            }
            i--;
        }
        if (denom_bigger == 1) { 
            numeratorindex++; 
            (resultnum->power)--;
        }
        
        // Now the main division loop. Note that there's two ways to terminate:
        // either we've filled the entire precision of the result word and are
        // forced to truncate our result, or our answer divides exactly. In the
        // second case, once we've exhausted the numerator's significant digits
        // and our temp word contains nothing but zeros, we can end early since
        // all subsequent iterations would contribute only zeros as well. Note
        // that special care will be taken to detect extra zeros at the end of
        // the result so that the sig_digs is recorded correctly. Also, we don't
        // round, we truncate, which doesn't minimize error.
        int nonzero = 1;
        while ((sigdigctr < (resultnum->precision)) && (nonzero == 1)) {
            // First run the subtraction loop.
            char current_digit = 0;
            int numer_bigger = 1;
            while (numer_bigger == 1) {
                // To subtract, first run a comparison to see if the numerator
                // is bigger. If it is, increment the counter and subtract. 
                i = ((denominator->sig_digs) - 1L);
                denom_bigger = 1;
                if (temp_word[(int)numeratorindex] > 0) { denom_bigger = 0; }
                while ((i >= 0L) && (denom_bigger == 1)) {
                    if ((denominator->digits[(int)((denominator->sig_digs) - 
                        i - 1L)]) > (temp_word[(int)((denominator->sig_digs) + 
                        numeratorindex - i)])) {
                        i = 0L;
                    }
                    else if ((denominator->digits[(int)((denominator->sig_digs) - 
                        i - 1L)]) < (temp_word[(int)((denominator->sig_digs) + 
                        numeratorindex - i)])) {
                        denom_bigger = 0;
                    }
                    else if (((denominator->digits[(int)((denominator->sig_digs) - 
                        i - 1L)]) == (temp_word[(int)((denominator->sig_digs) + 
                        numeratorindex - i)])) && (i == 0L)) {
                        denom_bigger = 0;
                    }
                    i--;
                }
                if (denom_bigger == 1) { 
                    numer_bigger = 0;
                }
                
                // Increment counter and perform subtraction loop.
                if (numer_bigger == 1) {
                    current_digit++;
                    for (i = 0L; i < (denominator->sig_digs); i++) {
                        temp_word[(int)((denominator->sig_digs) + 
                            numeratorindex - i)] -= (denominator->digits[
                            (int)((denominator->sig_digs) - i - 1L)]);
                        if ((temp_word[(int)((denominator->sig_digs) + 
                            numeratorindex - i)]) < 0) {
                            temp_word[(int)((denominator->sig_digs) + 
                                numeratorindex - i)] += 10L;
                            (temp_word[(int)((denominator->sig_digs) + 
                                numeratorindex - i - 1L)]) -= 1L;
                        }
                    }
                }
            }
            
            // If we're past all of the numerator's significant digits, run
            // zero detection on it to see if we can end early.
            if (sigdigctr > (numerator->sig_digs)) { // May only need to be >=
                long int zerocounter = 0L; 
                i = 0L;
                while ((i == zerocounter) && (i <= (denominator->sig_digs))) {
                    if ((temp_word[(int)(numeratorindex + i)]) < 1) { zerocounter++; }
                    i++;
                }
                if (zerocounter == ((denominator->sig_digs) + 1L)) { nonzero = 0; }
            }
            
            // Once we have obtained the proper digit in the result, save it.
            if (sigdigctr < resultnum->precision) {
                resultnum->digits[(int)sigdigctr] = current_digit;
            }
            sigdigctr++;
            numeratorindex++;
        }
        
        // Record the result's sig digs, taking care to detect trailing zeros.
        resultnum->sig_digs = sigdigctr;
        int trailingzeros = 1;
        long int zerocount = 0L;
        i = sigdigctr - 1L; 
        while (trailingzeros == 1) {
            if (resultnum->digits[(int)i] == '\0') {
                zerocount++;
            } else { trailingzeros = 0; }
            i--;
        }
        (resultnum->sig_digs) -= zerocount;
        free(temp_word);
        return 1;
    }
}

// A convenience wrapper that creates a temporary bignum out of the integer. 
// Since division is not commutative, two wrappers are given. Any problems 
// encountered in client functions are passed back up to the original caller.
__device__ int bignum_int_divide_gpu(bignum * resultnum, long int leftint, bignum * rightnum) {
    bignum_reset_gpu(resultnum);
    if (rightnum->sig_digs == 0L) { return 0; }
    else if (leftint == 0L) { return 1; }
    else {
        bignum * tempnum = bignum_init_gpu(resultnum->precision);
        if (tempnum == 0) { return 0; }
        if (bignum_set_int_gpu(tempnum, leftint) == 0) {
            bignum_clear_gpu(tempnum);
            return 0;
        }
        int retval = bignum_divide_gpu(resultnum, tempnum, rightnum);
        bignum_clear_gpu(tempnum);
        return retval;
    }
}

// A convenience wrapper that creates a temporary bignum out of the integer. 
// Since division is not commutative, two wrappers are given. Any problems 
// encountered in client functions are passed back up to the original caller.
__device__ int bignum_divide_int_gpu(bignum * resultnum, bignum * leftnum, long int rightint) {
    bignum_reset_gpu(resultnum);
    if (rightint == 0L) { return 0; }
    else if (leftnum->sig_digs == 0L) { return 1; }
    else {
        bignum * tempnum = bignum_init_gpu(resultnum->precision);
        if (tempnum == 0) { return 0; }
        if (bignum_set_int_gpu(tempnum, rightint) == 0) {
            bignum_clear_gpu(tempnum);
            return 0;
        }
        int retval = bignum_divide_gpu(resultnum, leftnum, tempnum);
        bignum_clear_gpu(tempnum);
        return retval;
    }
}
